
#include <hip/hip_runtime.h>
extern "C" {

#define TILE_WIDTH              (32)
#define KIRSCH_NUM_DIRS         (8)
#define KIRSCH_RADIUS           (1)
#define KIRSCH_WIDTH            (2*KIRSCH_RADIUS+1)
#define NUM_OUTPUT_CMAPS        (3)
#define NUM_OUTPUT_CHANNELS     (3)

__constant__ unsigned int CMAPS[NUM_OUTPUT_CMAPS][KIRSCH_NUM_DIRS][NUM_OUTPUT_CHANNELS];
__constant__ int KF[KIRSCH_NUM_DIRS][KIRSCH_WIDTH][KIRSCH_WIDTH];

__global__
void kirsch_filter(
    unsigned char * const I, unsigned char * const O,
    unsigned int const width, unsigned int height,
    unsigned int const thres,
    unsigned int const cmap,
    unsigned int const scale)
{
    int const row = blockDim.y*blockIdx.y + threadIdx.y;
    int const col = blockDim.x*blockIdx.x + threadIdx.x;

    if ((row >= height) || (col >= width)) return;

    /* Load input image tile into shared memory */
    __shared__ unsigned int I_s[TILE_WIDTH][TILE_WIDTH];
    I_s[threadIdx.y][threadIdx.x] = I[row*width + col];
    __syncthreads();

    /* Skip outer edge */
    if ((row < KIRSCH_RADIUS) || (row >= height-KIRSCH_RADIUS)
            || (col < KIRSCH_RADIUS) || (col >= width-KIRSCH_RADIUS))
        return;

    /* Compute directional derivatives, threshold and max direction */
    int max_deriv_d = -1;
    int max_deriv = thres;
    for (int d = 0; d < KIRSCH_NUM_DIRS; d++) {
        int deriv = 0;
        for (int i = -KIRSCH_RADIUS; i <= KIRSCH_RADIUS; i++) {
            for (int j = -KIRSCH_RADIUS; j <= KIRSCH_RADIUS; j++) {
                int const iy = threadIdx.y + i;
                int const jx = threadIdx.x + j;
                if ((iy >= 0) && (iy < TILE_WIDTH) &&
                        (jx >= 0) && (jx < TILE_WIDTH)) {
                    deriv += KF[d][i+KIRSCH_RADIUS][j+KIRSCH_RADIUS]
                                * I_s[iy][jx];
                } else {
                    deriv += KF[d][i+KIRSCH_RADIUS][j+KIRSCH_RADIUS]
                                * I[(row+i)*width+(col+i)];
                }
            }
        }

        if ((deriv > thres) && (deriv > max_deriv)) {
            max_deriv_d = d;
            max_deriv = deriv;
        }
    }

    /* Write output colour for max direction */
    if (max_deriv_d < 0) return;

    for (int c = 0; c < NUM_OUTPUT_CHANNELS; c++) {
        for (int sy = 0; sy < scale; sy++) {
            for (int sx = 0; sx < scale; sx++) {
                int const pos = ((row*scale+sy)*width*scale+(col*scale+sx))
                                    *NUM_OUTPUT_CHANNELS + c;
                O[pos] = CMAPS[cmap][max_deriv_d][c];
            }
        }
    }
}

} /* extern "C" */
